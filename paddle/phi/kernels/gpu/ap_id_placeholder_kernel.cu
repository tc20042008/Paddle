#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <mutex>
#include <unordered_map>
#include "glog/logging.h"
#include "jitify.hpp"  // NOLINT
#include "paddle/common/enforce.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_device_function.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/elementwise_base.h"
#include "paddle/phi/kernels/impl/activation_grad_impl.h"
#include "paddle/phi/kernels/impl/activation_impl.h"

#include "paddle/cinn/backends/nvrtc/nvrtc_util.h"
#include "paddle/cinn/runtime/cuda/cuda_module.h"

namespace ap {

const int kCUDAMaxCards{8};

/**
 * The CUDA module, helps to compile CUDA codes and fetch symbols.
 * Currently, it is a wrapper of NVRTC.
 */
class CUDAModule {
 public:
  enum class Kind {
    PTX = 0,
    CUBIN = 1,
  };

  CUDAModule(const std::string& data, Kind kind);

  void LaunchKernel(int device_id,
                    const std::string& func_name,
                    dim3 gridDim,
                    dim3 blockDim,
                    void** args,
                    size_t share_memory_size = 0,
                    hipStream_t stream = nullptr);

  //! Get a function.
  hipFunction_t GetFunction(int device_id, const std::string& func_name);

  //! Get a function by CudaGetDevice
  hipFunction_t GetFunction(const std::string& func_name);

  //! Get a global variable.
  hipDeviceptr_t GetGlobal(int device_id, const std::string& name, size_t nbytes);

  ~CUDAModule();

 private:
  //! The input data.
  std::string data_;
  //! Kind of the input.
  Kind kind_;
  //! To make parallel, we prepare one module for each card.
  std::vector<hipModule_t> module_per_card_{kCUDAMaxCards, nullptr};
  std::string cuda_source_;
  std::mutex mutex_;

  hipDevice_t device_;
  hipCtx_t context_;
  int num_devices_{0};
};

CUDAModule::CUDAModule(const std::string& data, Kind kind)
    : data_(data), kind_(kind) {
  PADDLE_ENFORCE_NE(
      data.empty(), true, phi::errors::PreconditionNotMet("data is is empty!"));

  hipGetDeviceCount(&num_devices_);
  PADDLE_ENFORCE_GT(
      num_devices_, 0, phi::errors::ResourceExhausted("No available devices!"));

  // TODO(Superjomn) Determine whether to initialize all the devices.
  int current_device_id;
  hipGetDevice(&current_device_id);
  hipSetDevice(current_device_id);
  hipDeviceGet(&device_, current_device_id);
  hipCtxGetCurrent(&context_);
  hipDevicePrimaryCtxRetain(&context_, device_);
  VLOG(5) << "Construct CUDAModule " << this
          << " in device: " << current_device_id;
}

void CUDAModule::LaunchKernel(int device_id,
                              const std::string& func_name,
                              dim3 gridDim,
                              dim3 blockDim,
                              void** args,
                              size_t share_memory_size,
                              hipStream_t stream) {
  VLOG(3) << "hipModuleLaunchKernel with func_name : " << func_name
          << ", gridDim.x:" << gridDim.x << ", gridDim.y:" << gridDim.y
          << ", gridDim.z:" << gridDim.z << ", blockDim.x:" << blockDim.x
          << ", blockDim.y:" << blockDim.y << ", blockDim.z:" << blockDim.z
          << ", share_memory_size:" << share_memory_size;
  auto function = GetFunction(device_id, func_name);
  PADDLE_ENFORCE_NOT_NULL(
      function,
      phi::errors::NotFound(
          "%s function not found on device %d.", func_name, device_id));
  CUDA_DRIVER_CALL(hipModuleLaunchKernel(function,
                                  gridDim.x,
                                  gridDim.y,
                                  gridDim.z,
                                  blockDim.x,
                                  blockDim.y,
                                  blockDim.z,
                                  share_memory_size,
                                  stream,
                                  args,
                                  nullptr));
}

hipFunction_t CUDAModule::GetFunction(const std::string& func_name) {
  int device_id;
  hipGetDevice(&device_id);
  return this->GetFunction(device_id, func_name);
}

hipFunction_t CUDAModule::GetFunction(int device_id,
                                   const std::string& func_name) {
  VLOG(5) << "GetFunction : " << func_name << " with device_id : " << device_id;
  if (!module_per_card_[device_id]) {
    std::lock_guard<std::mutex> lock(mutex_);
    // Compilation with parameters
    const size_t jit_num_options = 5;
    std::vector<hipJitOption> jit_options(jit_num_options);
    std::vector<void*> jit_opt_vals(jit_num_options);

    // set up size of compilation log buffer
    jit_options[0] = hipJitOptionErrorLogBufferSizeBytes;
    size_t log_buffer_size = 1024;
    jit_opt_vals[0] = reinterpret_cast<void*>(log_buffer_size);

    // set up pointer to the compilation log buffer
    jit_options[1] = hipJitOptionErrorLogBuffer;
    std::vector<char> log_buffer(log_buffer_size, '\0');
    jit_opt_vals[1] = log_buffer.data();

    int value = 1;
    // Specifies whether to create debug information in output (-g)
    jit_options[2] = hipJitOptionGenerateDebugInfo;
    jit_opt_vals[2] = reinterpret_cast<void*>(value);

    // Generate verbose log messages
    jit_options[3] = hipJitOptionLogVerbose;
    jit_opt_vals[3] = reinterpret_cast<void*>(value);

    // Generate line number information (-lineinfo)
    jit_options[4] = hipJitOptionGenerateLineInfo;
    jit_opt_vals[4] = reinterpret_cast<void*>(value);

    bool can_use_nvcc_compiler = false;
    if (can_use_nvcc_compiler) {
      CUDA_DRIVER_CALL(
          hipModuleLoad(&module_per_card_[device_id], data_.c_str()));
    } else {
      CUDA_DRIVER_CALL(hipModuleLoadDataEx(&module_per_card_[device_id],
                                          data_.c_str(),
                                          jit_num_options,
                                          jit_options.data(),
                                          jit_opt_vals.data()));
    }
  }

  hipFunction_t func;
  CUDA_DRIVER_CALL(hipModuleGetFunction(
      &func, module_per_card_[device_id], func_name.c_str()));
  return func;
}

hipDeviceptr_t CUDAModule::GetGlobal(int device_id,
                                  const std::string& name,
                                  size_t nbytes) {
  if (!module_per_card_[device_id]) {
    std::lock_guard<std::mutex> lock(mutex_);
    bool can_use_nvcc_compiler = false;
    if (can_use_nvcc_compiler) {
      CUDA_DRIVER_CALL(
          hipModuleLoad(&module_per_card_[device_id], data_.c_str()));
    } else {
      CUDA_DRIVER_CALL(
          hipModuleLoadData(&module_per_card_[device_id], data_.c_str()));
    }
  }

  size_t _nbytes;
  hipDeviceptr_t global;
  CUDA_DRIVER_CALL(hipModuleGetGlobal(
      &global, &_nbytes, module_per_card_[device_id], name.c_str()));
  return global;
}

CUDAModule::~CUDAModule() {
  for (int i = 0; i < module_per_card_.size(); i++) {
    auto* module = module_per_card_[i];
    if (module) {
      CUDA_CALL(hipSetDevice(i));
      CUDA_DRIVER_CALL(hipModuleUnload(module));
    }
  }
}

class HeaderGeneratorBase {
 public:
  virtual const size_t size() const = 0;
  virtual const std::vector<const char*>& headers() const = 0;
  virtual const std::vector<const char*>& include_names() const = 0;
};

class JitSafeHeaderGenerator : public HeaderGeneratorBase {
 public:
  static HeaderGeneratorBase& GetInstance();
  const size_t size() const;
  const std::vector<const char*>& headers() const override { return headers_; }
  const std::vector<const char*>& include_names() const override {
    return include_names_;
  }

 private:
  JitSafeHeaderGenerator();
  std::vector<const char*> headers_;
  std::vector<const char*> include_names_;
};

HeaderGeneratorBase& JitSafeHeaderGenerator::GetInstance() {
  static JitSafeHeaderGenerator instance;
  return instance;
}

const size_t JitSafeHeaderGenerator::size() const {
  PADDLE_ENFORCE_EQ(
      include_names_.size(),
      headers_.size(),
      phi::errors::InvalidArgument("Internal error in size of header files."));
  return include_names_.size();
}

JitSafeHeaderGenerator::JitSafeHeaderGenerator() {
  const auto& headers_map = ::jitify::detail::get_jitsafe_headers_map();
  for (auto& pair : headers_map) {
    include_names_.emplace_back(pair.first.data());
    headers_.emplace_back(pair.second.data());
  }
}

/**
 * An helper class to call NVRTC. Input CUDA device source code, get PTX string.
 */
class Compiler {
 public:
  Compiler();

  /**
   * Compile the \p code and get PTX string.
   * @param code The CUDA source code.
   * @param include_headers Whether to include the headers of CUDA and CINN
   * runtime modules.
   * @return Compiled PTX code string.
   */
  std::string operator()(const std::string& code, bool include_headers = true);

  /** Compile into cubin or not
   * @return Compile into cubin or not.
   */
  bool compile_to_cubin();

 private:
  /**
   * Get the directories of CUDA's header files.
   * @return list of header file directories.
   */
  std::vector<std::string> FindCUDAIncludePaths();

  /**
   * Get the directories of CINN runtime's header files.
   * @return list of header file directories.
   */
  std::vector<std::string> FindCINNRuntimeIncludePaths();

  /**
   * Compile CUDA source code and get PTX or CUBIN.
   * @param code source code string.
   * @return PTX or CUBIN string.
   */
  std::string CompileCudaSource(const std::string& code, bool include_headers);

  /**
   * whether to compile the source code into cubin, only works with cuda version
   * > 11.1
   */
  bool compile_to_cubin_{false};

  // compile with nvcc
  std::string CompileWithNvcc(const std::string&);

  // compile to ptx
  void CompileToPtx();
  // compile to cubin
  void CompileToCubin();
  std::string GetDeviceArch();

  std::string ReadFile(const std::string&, std::ios_base::openmode);

  std::string prefix_name_{""};
};

static bool TryLocatePath(const std::string& path) {
  struct stat st;
  return stat(path.c_str(), &st) == 0;
}

static std::vector<std::string> GetNvidiaAllIncludePath(
    const std::string& nvidia_package_dir) {
  std::vector<std::string> include_paths;
  const std::string delimiter = "/";
  // Expand this list if necessary.
  const std::vector<std::string> sub_modules = {"cublas",
                                                "cudnn",
                                                "cufft",
                                                "cusparse",
                                                "cusolver",
                                                "cuda_nvrtc",
                                                "hiprand",
                                                "cuda_runtime"};
  for (auto& sub_module : sub_modules) {
    std::string path =
        nvidia_package_dir + delimiter + sub_module + delimiter + "include";
    include_paths.push_back(path);
  }
  return include_paths;
}

std::string Compiler::operator()(const std::string& code,
                                 bool include_headers) {
  bool can_use_nvcc_compiler = false;
  if (can_use_nvcc_compiler) {
    return CompileWithNvcc(code);
  }
  return CompileCudaSource(code, include_headers);
}

Compiler::Compiler() {
  // Do nothing.
}

bool Compiler::compile_to_cubin() { return compile_to_cubin_; }

std::vector<std::string> Compiler::FindCUDAIncludePaths() {
  const std::string delimiter = "/";
  std::string cuda_include_path;
  const char* cuda_path_env = std::getenv("CUDA_PATH");
  if (cuda_path_env != nullptr) {
    cuda_include_path += cuda_path_env;
    cuda_include_path += delimiter + "include";
    VLOG(4) << "FindCUDAIncludePaths from CUDA_PATH: " << cuda_include_path;
    return {cuda_include_path};
  }

#if defined(__linux__)

  cuda_include_path = "/usr/local/cuda/include";
  if (TryLocatePath(cuda_include_path)) {
    VLOG(4) << "FindCUDAIncludePaths from " << cuda_include_path;
    return {cuda_include_path};
  }
#endif
  std::stringstream ss;
  ss << "Cannot find cuda include path."
     << "CUDA_PATH is not set or CUDA is not installed in the default "
        "installation path."
     << "In other than linux, it is necessary to set CUDA_PATH.";
  PADDLE_THROW(phi::errors::Fatal(ss.str()));
  return {cuda_include_path};
}

std::vector<std::string> Compiler::FindCINNRuntimeIncludePaths() { return {}; }

std::string Join(const std::vector<std::string>& strs, const std::string& sep) {
  std::string ret;
  int i = 0;
  for (const auto& str : strs) {
    if (i++ > 0) {
      ret += sep;
    }
    ret += str;
  }
  return ret;
}

std::string UniqName(const std::string& prefix) {
  const size_t seq_no = [&] {
    static std::mutex mutex;
    static std::unordered_map<std::string, size_t> prefix2seq_no;
    static std::unique_lock<std::mutex> lock(mutex);
    return prefix2seq_no[prefix]++;
  }();
  return prefix + "_" + std::to_string(seq_no);
}

std::string Compiler::CompileCudaSource(const std::string& code,
                                        bool include_headers) {
  const auto& header_gen = JitSafeHeaderGenerator::GetInstance();
  std::vector<std::string> compile_options;
  std::vector<const char*> param_cstrings{};
  hiprtcProgram prog;
  std::string cc = "30";
  int major, minor;
  hipError_t e1 =
      hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, 0);
  hipError_t e2 =
      hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, 0);

  if (e1 == hipSuccess && e2 == hipSuccess) {
    cc = std::to_string(major) + std::to_string(minor);
  } else {
    LOG(WARNING) << "cannot detect compute capability from your device, "
                 << "fall back to compute_30.";
  }
  if (compile_to_cubin_) {
    compile_options.push_back("-arch=sm_" + cc);
    std::string enable_fmad = "false";
    compile_options.push_back("--fmad=" + enable_fmad);
  } else {
    compile_options.push_back("-arch=compute_" + cc);
  }
  compile_options.push_back("-std=c++14");
  compile_options.push_back("-default-device");

  if (include_headers) {  // prepare include headers
    auto cuda_headers = FindCUDAIncludePaths();
    auto cinn_headers = FindCINNRuntimeIncludePaths();
    std::vector<std::string> include_paths;
    for (auto& header : cuda_headers) {
      VLOG(5) << "add include-path: " << header;
      include_paths.push_back("--include-path=" + header);
    }
    for (auto& header : cinn_headers) {
      include_paths.push_back("--include-path=" + header);
    }
    compile_options.insert(
        std::end(compile_options), include_paths.begin(), include_paths.end());
  }

  for (const auto& option : compile_options) {
    param_cstrings.push_back(option.c_str());
  }
  VLOG(3) << "compile options: " << Join(compile_options, " ");
  NVRTC_CALL(hiprtcCreateProgram(&prog,
                                code.c_str(),
                                nullptr,
                                header_gen.size(),
                                header_gen.headers().data(),
                                header_gen.include_names().data()));
  hiprtcResult compile_res =
      hiprtcCompileProgram(prog, param_cstrings.size(), param_cstrings.data());

  const auto& GetLog = [&]() {
    size_t log_size = 0;
    NVRTC_CALL(hiprtcGetProgramLogSize(prog, &log_size));
    std::string log;
    log.resize(log_size);
    NVRTC_CALL(hiprtcGetProgramLog(prog, &log[0]));
    return log;
  };

  {  // get log
    PADDLE_ENFORCE_EQ(
        compile_res,
        HIPRTC_SUCCESS,
        phi::errors::Fatal("NVRTC compilation failed. "
                           "\n================[code]================\n" +
                           code + "\n================[log]================\n" +
                           GetLog()));
  }

  size_t size;
  std::string data;
  if (compile_to_cubin_) {
    NVRTC_CALL(hiprtcGetBitcodeSize(prog, &size));
    data.resize(size);
    NVRTC_CALL(hiprtcGetBitcode(prog, &data[0]));
  } else {
    NVRTC_CALL(hiprtcGetCodeSize(prog, &size));
    data.resize(size);
    NVRTC_CALL(hiprtcGetCode(prog, &data[0]));
  }

  NVRTC_CALL(hiprtcDestroyProgram(&prog));
  return data;
}

std::string Compiler::CompileWithNvcc(const std::string& cuda_c) {
  // read dir source
  std::string dir = "./source";
  if (access(dir.c_str(), 0) == -1) {
    CHECK(mkdir(dir.c_str(), 7) != -1) << "Fail to mkdir " << dir;
  }

  // get unique prefix name
  prefix_name_ = dir + "/" + UniqName("rtc_tmp");

  auto cuda_c_file = prefix_name_ + ".cu";
  std::ofstream ofs(cuda_c_file, std::ios::out);
  CHECK(ofs.is_open()) << "Fail to open file " << cuda_c_file;
  ofs << cuda_c;
  ofs.close();

  CompileToPtx();
  CompileToCubin();

  return prefix_name_ + ".cubin";
}

// std::string Compiler::GetPtx() { return ReadFile(prefix_name_ + ".ptx",
// std::ios::in); }

void Compiler::CompileToPtx() {
  std::vector<std::string> include_dir = {};
  std::string include_dir_str = "";
  for (auto dir : include_dir) {
    if (include_dir_str.empty()) {
      include_dir_str = dir;
    } else {
      include_dir_str += ":" + dir;
    }
  }
  const std::string FLAGS_cinn_nvcc_cmd_path = "/usr/local/cuda/bin";
  std::string options = std::string("export PATH=") + FLAGS_cinn_nvcc_cmd_path +
                        std::string(":$PATH && nvcc -std=c++14 --ptx -O3 -I ") +
                        include_dir_str;
  options += " -arch=" + GetDeviceArch();
  options += " -o " + prefix_name_ + ".ptx";
  options += " " + prefix_name_ + ".cu";

  VLOG(2) << "Nvcc Compile Options : " << options;
  CHECK(system(options.c_str()) == 0) << options;
}

void Compiler::CompileToCubin() {
  const std::string FLAGS_cinn_nvcc_cmd_path = "/usr/local/cuda/bin";
  std::string options = std::string("export PATH=") + FLAGS_cinn_nvcc_cmd_path +
                        std::string(":$PATH && nvcc --cubin -O3");
  options += " -arch=" + GetDeviceArch();
  options += " -o " + prefix_name_ + ".cubin";
  options += " " + prefix_name_ + ".ptx";

  VLOG(2) << "Nvcc Compile Options : " << options;
  CHECK(system(options.c_str()) == 0) << options;
}

std::string Compiler::GetDeviceArch() {
  int major = 0, minor = 0;
  if (hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, 0) ==
          hipSuccess &&
      hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, 0) ==
          hipSuccess) {
    return "sm_" + std::to_string(major) + std::to_string(minor);
  } else {
    LOG(WARNING) << "cannot detect compute capability from your device, "
                 << "fall back to compute_30.";
    return "sm_30";
  }
}

std::string Compiler::ReadFile(const std::string& file_name,
                               std::ios_base::openmode mode) {
  // open cubin file
  std::ifstream ifs(file_name, mode);
  CHECK(ifs.is_open()) << "Fail to open file " << file_name;
  ifs.seekg(std::ios::end);
  auto len = ifs.tellg();
  ifs.seekg(0);

  // read cubin file
  std::string file_data(len, ' ');
  ifs.read(&file_data[0], len);
  ifs.close();
  return std::move(file_data);
}

}  // namespace ap

namespace cinn::runtime::cuda {

template <typename T, typename Context>
void ApIdPlaceholderKernel(const Context& dev_ctx,
                           const phi::DenseTensor& x,
                           phi::DenseTensor* out) {
  auto generate_ptx = [] {
    ap::Compiler compiler;

    std::string source_code = R"(
  #include <cstdint>
  #define CINN_WITH_CUDA

  extern "C" __global__
  void relu(const float* input, const int num, float* output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx < num) {
      output[idx] = input[idx] > 0 ? input[idx] : 0;
    }
  }
  )";

    auto ptx = compiler(source_code);
    CHECK(!ptx.empty());
    return ptx;
  };

  auto ptx = generate_ptx();

  ap::CUDAModule cuda_module(ptx, ap::CUDAModule::Kind::PTX);
  int size = x.numel();
  dim3 blocks_per_grid(1);
  dim3 threads_per_block(100);
  const void* x_data = x.data();
  void* out_data = out->data();
  void* args[] = {&x_data, &size, &out_data};
  cuda_module.LaunchKernel(0, "relu", blocks_per_grid, threads_per_block, args);
}

}  // namespace cinn::runtime::cuda

namespace phi {

template <typename T, typename Context>
void ApIdPlaceholderKernel(const Context& dev_ctx,
                           const DenseTensor& x,
                           DenseTensor* out) {
  dev_ctx.template Alloc<T>(out);
  cinn::runtime::cuda::ApIdPlaceholderKernel<T, Context>(dev_ctx, x, out);
}

}  // namespace phi

#ifdef PADDLE_WITH_HIP
PD_REGISTER_KERNEL(ap_id_placeholder,
                   GPU,
                   ALL_LAYOUT,
                   phi::ApIdPlaceholderKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
#else
PD_REGISTER_KERNEL(ap_id_placeholder,
                   GPU,
                   ALL_LAYOUT,
                   phi::ApIdPlaceholderKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
#endif
