#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include <mutex>
#include <unordered_map>
#include "glog/logging.h"
#include "jitify.hpp"  // NOLINT
#include "paddle/common/enforce.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_device_function.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/elementwise_base.h"
#include "paddle/phi/kernels/impl/activation_grad_impl.h"
#include "paddle/phi/kernels/impl/activation_impl.h"

#include "paddle/cinn/backends/nvrtc/nvrtc_util.h"
#include "paddle/cinn/runtime/cuda/cuda_module.h"
#include "paddle/phi/kernels/gpu/ap_cuda_jit_util.h"

namespace ap {

template <typename T, typename Context>
void ApUnaryKernel(const Context& dev_ctx,
                   const phi::DenseTensor& x,
                   phi::DenseTensor* out) {
  auto generate_ptx = [] {
    ap::Compiler compiler;

    std::string source_code = R"(
  #include <cstdint>
  #define CINN_WITH_CUDA

  extern "C" __global__
  void relu(const float* input, const int num, float* output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx < num) {
      output[idx] = input[idx] > 0 ? input[idx] : 0;
    }
  }
  )";
    LOG(ERROR) << "\n" << source_code;
    auto ptx = compiler(source_code);
    CHECK(!ptx.empty());
    return ptx;
  };

  auto ptx = generate_ptx();

  ap::CUDAModule cuda_module(ptx, ap::CUDAModule::Kind::PTX);
  int size = x.numel();
  dim3 blocks_per_grid(1);
  dim3 threads_per_block(100);
  const void* x_data = x.data();
  void* out_data = out->data();
  void* args[] = {&x_data, &size, &out_data};
  cuda_module.LaunchKernel(0, "relu", blocks_per_grid, threads_per_block, args);
}

}  // namespace ap

namespace phi {

template <typename T, typename Context>
void ApUnaryKernel(const Context& dev_ctx,
                   const std::vector<const DenseTensor*>& xs,
                   int num_outputs,
                   std::vector<DenseTensor*> outs) {
  PADDLE_ENFORCE_GT(
      xs.size(),
      0,
      phi::errors::InvalidArgument(
          "At least 1 input is required. current number out uts: // %d",
          xs.size()));
  PADDLE_ENFORCE_GT(
      outs.size(),
      0,
      phi::errors::InvalidArgument(
          "num_outputs must be greater than 1. current _outputs: // %d",
          outs.size()));
  for (auto* out : outs) {
    dev_ctx.template Alloc<T>(out);
  }
  ap::ApUnaryKernel<T, Context>(dev_ctx, *xs[0], outs[0]);
}

}  // namespace phi

#ifdef PADDLE_WITH_HIP
PD_REGISTER_KERNEL(ap_unary,
                   GPU,
                   ALL_LAYOUT,
                   phi::ApUnaryKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
#else
PD_REGISTER_KERNEL(ap_unary,
                   GPU,
                   ALL_LAYOUT,
                   phi::ApUnaryKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
#endif
